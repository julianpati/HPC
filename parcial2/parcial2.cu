#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "someDefinitions.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>

// Error handling macro
#define CUDA_CHECK(call) \
    if((call) != hipSuccess) { \
        hipError_t err = hipGetLastError(); \
        printf("CUDA error calling, code is %d\n", err);}



__global__ void matrixMulKernel(double *d_a, double *d_b, double *d_c, int height, int width_a, int width_b) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < height && col < width_b) {
    double p_result = 0;
    for (int k = 0; k < width_a; k++) {
      p_result += d_a[row * width_a + k] * d_b[k * width_b + col];
    }
    d_c[row * width_b + col] = p_result;
  }
}

void cudaCall(int f1, int c1, int f2, double* M1, double* M2, double* M3){
	double *d_M, *d_N, *d_P;
	hipMalloc(&d_M, sizeof(double) * f1 * c1);
	hipMalloc(&d_N, sizeof(double) * c1 * f2);
	hipMalloc(&d_P, sizeof(double) * f1 * f2);
  
	hipError_t error = hipSuccess;

	error = hipMemcpy(d_M, M1, f1 * c1 * sizeof(double), hipMemcpyHostToDevice);
	if(error != hipSuccess){
		printf("Error copiando datos a d_M");
		exit(0);
	}

	error = hipMemcpy(d_N, M2, c1 * f2 * sizeof(double), hipMemcpyHostToDevice);
	if(error != hipSuccess){
		printf("Error copiando datos a d_N");
		exit(0);
	}

	int blockSize = 32;
	dim3 dimBlock(blockSize,blockSize,1);
	dim3 dimGrid(ceil(f1/double(blockSize)),ceil(f2/double(blockSize)),1);
	matrixMulKernel<<<dimGrid,dimBlock>>>(d_M,d_N,d_P, f1, c1, f2);
	hipDeviceSynchronize();
	hipMemcpy(M3,d_P, f1 * f2 * sizeof(double),hipMemcpyDeviceToHost);
	hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P);
	//Fin multiplicacion con GPU
}
